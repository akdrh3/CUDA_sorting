#include "gpu_util.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
extern "C" {
#include "util.h"
}

int main() {
    char file_name[256];
    printf("Enter the file name: ");
    scanf("%255s", file_name);

    uint64_t size_of_array = count_size_of_file(file_name);
    printf("Number of integers in the file : %lu\n", size_of_array);

    int *number_array = NULL;
    read_from_file(file_name, &number_array, size_of_array);
    printf("Last element: %d\n", number_array[size_of_array - 1]);

    free(number_array);

    return 0;
}