#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>

__device__ void swap(int *a, int *b)
{
    int temp = *a;
    *a = *b;
    *b = temp;
}

__device__ int partition(int *arr, int left, int right)
{
    int pivot = arr[right];
    int i = left - 1;

    for (int j = left; j <= right - 1; ++j)
    {
        if (arr[j] <= pivot)
        {
            ++i;
            swap(&arr[i], &arr[j]);
        }
    }
    swap(&arr[i + 1], &arr[right]);
    return (i + 1);
}

__global__ void quickSort(int *arr, int left, int right)
{
    // Stack for storing left and right indices
    int stack[1024];

    // Initialize stack
    int top = -1;
    stack[++top] = left;
    stack[++top] = right;

    // Pop from stack and push sub-arrays
    while (top >= 0)
    {
        right = stack[top--];
        left = stack[top--];

        int p = partition(arr, left, right);

        // If there are elements on the left side of the pivot, push left side to stack
        if (p - 1 > left)
        {
            stack[++top] = left;
            stack[++top] = p - 1;
        }

        // If there are elements on the right side of the pivot, push right side to stack
        if (p + 1 < right)
        {
            stack[++top] = p + 1;
            stack[++top] = right;
        }
    }
}

void performQuickSortAndMeasureTime(const std::string &filename)
{
    // Read numbers from file
    std::ifstream inputFile(filename);
    if (!inputFile)
    {
        std::cerr << "Failed to open the file " << filename << std::endl;
        return;
    }

    std::vector<int> vec;
    int number;
    while (inputFile >> number)
    {
        vec.push_back(number);
    }
    inputFile.close();
    std::cout << "starting quicksort ... " << std::endl;
    int n = vec.size();
    int *d_arr;
    hipMalloc(&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, vec.data(), n * sizeof(int), hipMemcpyHostToDevice);

    // Measure time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    quickSort<<<1, 1>>>(d_arr, 0, n - 1);
    hipEventRecord(stop);
    hipDeviceSynchronize();

    hipMemcpy(vec.data(), d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_arr);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Time taken to quick sort " << n << " elements: " << milliseconds << " s" << std::endl;

    // // Optionally, print the sorted array
    // std::cout << "Sorted output: ";
    // for (int i = 0; i < n; ++i)
    // {
    //     std::cout << vec[i] << " ";
    // }
    // std::cout << std::endl;
}

int main()
{
    performQuickSortAndMeasureTime("numbers.txt");
    return 0;
}