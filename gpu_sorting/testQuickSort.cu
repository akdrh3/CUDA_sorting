#include "hip/hip_runtime.h"
#include "gpu_util.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
extern "C" {
#include "util.h"
}

__device__ void swap(int *a, int *b) {
    int tmp = 0;
    tmp = *a;
    *a = *b;
    *b = tmp;
}

__device__ void gpu_print_array(int *int_array, int64_t array_size) {
    for (int64_t i = 0; i < array_size; ++i) {
        printf("%d ", int_array[i]);
    }
    printf("\n");
}

__device__ int64_t partition(int *arr, int64_t low, int64_t high) {
    int pivot = arr[high];
    int64_t i = low - 1;

    for (int64_t j = low; j < high; j++) {
        if (arr[j] < pivot) {
            i = i + 1;
            swap(&arr[i], &arr[j]);
        }
    }
    swap(&arr[i + 1], &arr[high]);

    return i + 1;
}

__global__ void quickSortKernel(int *arr, int64_t size) {
    // Dynamically allocate stack based on the size of the array using shared memory
    extern __shared__ int64_t stack[];
    int64_t top = -1;

    // Push initial low and high indexes
    top++;
    stack[top] = 0;
    top++;
    stack[top] = size - 1;

    // Keep popping from the stack while it's not empty
    while (top >= 0) {
        int64_t high = stack[top];
        top--;
        int64_t low = stack[top];
        top--;

        // Partition the array
        int64_t pi = partition(arr, low, high);

        // Push left side to stack if needed
        if (pi - 1 > low) {
            top++;
            stack[top] = low;
            top++;
            stack[top] = pi - 1;
        }

        // Push right side to stack if needed
        if (pi + 1 < high) {
            top++;
            stack[top] = pi + 1;
            top++;
            stack[top] = high;
        }
    }
}

int main() {
    // Read the numbers from the file into an array in CPU memory.
    char file_name[256];
    printf("Enter the file name: \n");
    scanf("%255s", file_name);

    uint64_t size_of_array = count_size_of_file(file_name);
    printf("Number of integers in the file : %lu\n", size_of_array);

    int *number_array = NULL;
    read_from_file(file_name, &number_array, size_of_array);

    // Allocate memory on the GPU.
    int *gpu_number_array = NULL;
    HANDLE_ERROR(hipMallocManaged(&gpu_number_array, sizeof(int) * size_of_array));

    // Copy the array from CPU memory to GPU memory.
    memcpy(gpu_number_array, number_array, sizeof(int) * size_of_array);

    // Thread options array
    int threads_options[5] = {1, 256, 512, 768, 1024};

    // Iterate through each thread configuration
    for (int i = 0; i < 5; ++i) {
        // Re-allocate memory on the GPU for each iteration
        HANDLE_ERROR(hipMallocManaged(&gpu_number_array, sizeof(int) * size_of_array));

        // Copy the array from CPU memory to GPU memory.
        memcpy(gpu_number_array, number_array, sizeof(int) * size_of_array);

        int threadsPerBlock = threads_options[i];
        int blocksPerGrid = (size_of_array + threadsPerBlock - 1) / threadsPerBlock;

        // Print current configuration
        printf("Running QuickSort with %d threads per block...\n", threadsPerBlock);

        // Start timer
        hipEvent_t start, stop;
        cuda_timer_start(&start, &stop);

        // Launch kernel with different thread configurations
        quickSortKernel<<<blocksPerGrid, threadsPerBlock, sizeof(int64_t) * size_of_array>>>(gpu_number_array, size_of_array);
        HANDLE_ERROR(hipDeviceSynchronize());

        // Stop timer
        double gpu_sort_time = cuda_timer_stop(start, stop);
        double gpu_sort_time_sec = gpu_sort_time / 1000.0;

        // Print elapsed time for the current configuration
        printf("Time elapsed for %d threads per block: %lf s\n\n", threadsPerBlock, gpu_sort_time_sec);

        // Optionally print sorted array
        print_array(gpu_number_array, size_of_array);

        // Free GPU memory for this iteration
        HANDLE_ERROR(hipFree(gpu_number_array));
    }

    // Free the host memory
    free(number_array);

    return 0;
}
