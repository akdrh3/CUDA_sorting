#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>

// CUDA kernel to merge two sorted subarrays
__global__ void merge(int *arr, int *temp, int left, int mid, int right)
{
    int i = left, j = mid + 1, k = left;

    while (i <= mid && j <= right)
    {
        if (arr[i] <= arr[j])
        {
            temp[k++] = arr[i++];
        }
        else
        {
            temp[k++] = arr[j++];
        }
    }

    while (i <= mid)
    {
        temp[k++] = arr[i++];
    }

    while (j <= right)
    {
        temp[k++] = arr[j++];
    }

    for (int i = left; i <= right; i++)
    {
        arr[i] = temp[i];
    }
}

// Function to recursively split and merge the array on the host
void mergeSortHost(int *d_arr, int *d_temp, int left, int right)
{
    if (left < right)
    {
        int mid = (left + right) / 2;

        // Recursively sort the left and right halves
        mergeSortHost(d_arr, d_temp, left, mid);
        mergeSortHost(d_arr, d_temp, mid + 1, right);

        // Merge the sorted halves on the device
        merge<<<1, 1>>>(d_arr, d_temp, left, mid, right);
        hipDeviceSynchronize();
    }
}

// CUDA kernel for quick sort partition
__device__ int partition(int *arr, int low, int high)
{
    int pivot = arr[high];
    int i = low - 1;
    for (int j = low; j < high; j++)
    {
        if (arr[j] <= pivot)
        {
            i++;
            int temp = arr[i];
            arr[i] = arr[j];
            arr[j] = temp;
        }
    }
    int temp = arr[i + 1];
    arr[i + 1] = arr[high];
    arr[high] = temp;
    return i + 1;
}

// CUDA kernel for quick sort
__global__ void quickSort(int *arr, int low, int high)
{
    if (low < high)
    {
        int pi = partition(arr, low, high);

        quickSort<<<1, 1>>>(arr, low, pi - 1);
        quickSort<<<1, 1>>>(arr, pi + 1, high);
        hipDeviceSynchronize();
    }
}

// Function to perform sorting and measure time
void performSortAndMeasureTime(const std::string &filename)
{
    std::ifstream inputFile(filename);
    if (!inputFile)
    {
        std::cerr << "Failed to open the file " << filename << std::endl;
        return;
    }

    std::vector<int> vec;
    int number;
    while (inputFile >> number)
    {
        vec.push_back(number);
    }
    inputFile.close();

    int n = vec.size();
    int *d_arr;
    int *d_temp;
    hipMalloc(&d_arr, n * sizeof(int));
    hipMalloc(&d_temp, n * sizeof(int));
    hipMemcpy(d_arr, vec.data(), n * sizeof(int), hipMemcpyHostToDevice);

    // Measure merge sort time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    mergeSortHost(d_arr, d_temp, 0, n - 1);
    hipEventRecord(stop);

    hipMemcpy(vec.data(), d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float mergeSortMilliseconds = 0;
    hipEventElapsedTime(&mergeSortMilliseconds, start, stop);

    std::cout << "Merge sort time for " << filename << ": " << mergeSortMilliseconds << " ms" << std::endl;

    hipMemcpy(d_arr, vec.data(), n * sizeof(int), hipMemcpyHostToDevice);

    // Measure quick sort time
    hipEventRecord(start);
    quickSort<<<1, 1>>>(d_arr, 0, n - 1);
    hipEventRecord(stop);

    hipMemcpy(vec.data(), d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float quickSortMilliseconds = 0;
    hipEventElapsedTime(&quickSortMilliseconds, start, stop);

    std::cout << "Quick sort time for " << filename << ": " << quickSortMilliseconds << " ms" << std::endl;

    hipFree(d_arr);
    hipFree(d_temp);
}

int main()
{
    std::vector<std::string> filenames = {
        "oneMillionNum.txt",
        "twoMillionNum.txt",
        "fourMillionNum.txt",
        "eightMillionNum.txt",
        "sxtnMillionNum.txt",
        "thrtytwMillionNum.txt"};

    for (const auto &filename : filenames)
    {
        performSortAndMeasureTime(filename);
    }

    return 0;
}